#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <ctime>

// Kernel de redução: cada bloco soma parte do vetor e armazena o resultado em memória compartilhada.
// No final, o primeiro thread do bloco escreve o resultado parcial no vetor de saída.
__global__ void reduceKernel(const float* __restrict__ d_in, float* __restrict__ d_out, size_t n) {
    extern __shared__ float sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    float mySum = 0.0f;

    // Carrega dados na memória compartilhada, realizando o "unrolling"
    if (i < n) mySum = d_in[i];
    if (i + blockDim.x < n) mySum += d_in[i + blockDim.x];
    sdata[tid] = mySum;
    __syncthreads();

    // Redução na memória compartilhada
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // O primeiro thread do bloco escreve o resultado parcial
    if (tid == 0) d_out[blockIdx.x] = sdata[0];
}

// Função host para realizar a redução múltipla até restar apenas um valor
float gpuReduceSum(const float* d_in, size_t n) {
    // Definir tamanho do bloco
    int blockSize = 256;  
    int gridSize = (int)((n + blockSize * 2 - 1) / (blockSize * 2));

    // Alocar memória para resultados parciais
    float *d_intermediate, *d_final;
    hipMalloc(&d_intermediate, sizeof(float) * gridSize);
    hipMalloc(&d_final, sizeof(float));

    size_t sharedMemSize = blockSize * sizeof(float);
    // Primeira redução
    reduceKernel<<<gridSize, blockSize, sharedMemSize>>>(d_in, d_intermediate, n);

    // Reduzir até sobrar um bloco
    int s = gridSize;
    while (s > 1) {
        int nextGridSize = (s + blockSize * 2 - 1) / (blockSize * 2);
        reduceKernel<<<nextGridSize, blockSize, sharedMemSize>>>(d_intermediate, d_intermediate, s);
        s = nextGridSize;
    }

    // Copiar o resultado final
    hipMemcpy(d_final, d_intermediate, sizeof(float), hipMemcpyDeviceToDevice);

    float h_result;
    hipMemcpy(&h_result, d_final, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_intermediate);
    hipFree(d_final);

    return h_result;
}

int main() {
    srand((unsigned)time(nullptr));

    size_t n = 100000001; // tamanho do vetor

    // Alocar vetor na CPU
    float* h_vec = (float*)malloc(n * sizeof(float));

    // Preencher vetor com valores aleatórios
    for (size_t i = 0; i < n; i++) {
        h_vec[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // Alocar e copiar vetor para GPU
    float* d_vec;
    hipMalloc(&d_vec, n * sizeof(float));
    hipMemcpy(d_vec, h_vec, n * sizeof(float), hipMemcpyHostToDevice);

    // Medição do tempo de execução
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Realizar soma na GPU
    float soma = gpuReduceSum(d_vec, n);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calcular o tempo de execução
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Imprime resultado
    std::cout << "Soma do vetor: " << soma << std::endl;
    std::cout << "Tempo de execução: " << milliseconds << " ms" << std::endl;

    // Liberar memória
    hipFree(d_vec);
    free(h_vec);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}